#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "module.cuh"
#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calc_2d(int * input){
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bx = blockDim.x;
    int by = blockDim.y;
    int block_offsety = bx*tiy;
    int col_offsetx = bx*by*blockIdx.x;
    int row_offsety = bx*by*gridDim.x*blockIdx.y;
    int gid = tix+block_offsety+col_offsetx+row_offsety;
    printf("gid: %d, value: %d\n",gid,input[gid]);
}

__global__ void square_arr_kernel(float *a, unsigned int numElements){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numElements)
        a[i] = a[i] * a[i];
}

void test(float *a, unsigned int numElements){
    size_t size = numElements * sizeof(float);
    float* a_device;

    hipMalloc((void**)&a_device, size);
    hipMemcpy(a_device, a, size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (numElements + block_size - 1) / block_size;

    square_arr_kernel <<<grid_size, block_size>>> (a_device, numElements);

    hipMemcpy(a, a_device, size, hipMemcpyDeviceToHost);
    hipFree(a_device);

}