#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "common.cuh"
#include "common.hpp"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void sum_arrays_1Dgrid_1Dblock(float * a, float * b, float * c, int nx) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    if (gid<nx)
        c[gid] = a[gid] + b[gid];
}

__global__ void sum_arrays_2Dgrid_2Dblock(float * a, float * b, float * c, int nx, int ny) {
    int gidx = blockIdx.x*blockDim.x + threadIdx.x;
    int gidy = blockIdx.y*blockDim.y + threadIdx.y;

    int gid = gidy*nx + ny;

    if (gidx < nx && gidy < ny)
        c[gid] = a[gid] + b[gid];
}

void run_sum_array_1d(int argc, char** argv) {

}

void run_sum_array_2d(int argc, char** argv) {

}
