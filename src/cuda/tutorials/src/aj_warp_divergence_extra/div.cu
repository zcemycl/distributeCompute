#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"


__global__ void divergence_code() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    float a,b;
    a = b = 0;

    if (gid % 2 == 0){
        a = 100.0;
        b = 50.0;
    } else {
        a = 200.0;
        b = 75.0;
    }

    // printf("a: %f, b: %f, gid: %d", a, b, gid);
}


int main(){
    printf("\n---------------------WARP DIVERGENCE EXAMPLE--------------------\n\n");

    int size = 1<<22;

    dim3 block(128);
    dim3 grid((size+block.x-1)/block.x);

    divergence_code <<<grid,block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset(); 

    return 0;
}
