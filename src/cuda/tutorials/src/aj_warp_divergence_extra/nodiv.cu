#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"


__global__ void no_divergence_code() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    float a,b;
    a = b = 0;

    // printf("a: %f, b: %f, gid: %d", a, b, gid);
}


int main(){
    printf("\n---------------------WARP DIVERGENCE EXAMPLE--------------------\n\n");

    int size = 1<<22;

    dim3 block(128);
    dim3 grid((size+block.x-1)/block.x);

    no_divergence_code <<<grid,block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset(); 

    return 0;
}
