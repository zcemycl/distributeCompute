#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "ai_warp_details/module.cuh"


__global__ void warp_details() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x
        + blockDim.x*gridDim.x*blockIdx.y;
    int local_warp_id = threadIdx.x/32;
    int global_block_id = blockIdx.x + gridDim.x*blockIdx.y;
    int warp_x = blockDim.x/32 + 1;
    int global_warp_id = threadIdx.x/32 + warp_x*blockIdx.x 
        + warp_x*gridDim.x*blockIdx.y;
}

void print_warp_details(){

}