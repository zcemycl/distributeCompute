#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "ac_blockgrid/module.cuh"

__global__ void print_blockgrid() {
    printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, blockDim.z : %d, gridDim.x : %d, gridDim.y : %d, gridDim.z : %d\n",
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

void play_blockgrid(){
    int nx = 16, ny = 16;
    dim3 block(8,8);
    dim3 grid(nx/block.x, ny/block.y);

    print_blockgrid <<<grid,block>>> ();

    hipDeviceSynchronize();
    hipDeviceReset();
}