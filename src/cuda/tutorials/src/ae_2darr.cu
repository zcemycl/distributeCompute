#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ae_2darr/module.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void unique_gid_calc_2d(int * data){
    int tid = threadIdx.x;
    int block_offset = blockIdx.x * blockDim.x;
    int row_offset = blockDim.x * gridDim.x * blockIdx.y;

    int gid = row_offset + block_offset + tid;
    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
        blockIdx.x, blockIdx.y, tid, gid, data[gid]);

}

__global__ void unique_gid_calc_2d_2d(int * data){
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_threads_in_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_in_block;
    int num_threads_in_row = num_threads_in_block * gridDim.x;
    int row_offset = num_threads_in_row * blockIdx.y;

    int gid = row_offset + block_offset + tid;
    printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d - data : %d \n",
        blockIdx.x, blockIdx.y, tid, gid, data[gid]);

}

void access_2d_arr() {
    int array_size = 16;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23,9,4,53,65,12,1,33,22,43,56,4,76,81,94,32}; // host data

    int * d_data; // device data
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(4);
    dim3 grid(2,2);

    unique_gid_calc_2d <<<grid,block>>> (d_data);
    hipDeviceSynchronize();
    
    dim3 block2(2,2);
    std::cout << "Grid: 2x2, Block: 2x2 ... " << std::endl;
    unique_gid_calc_2d_2d <<<grid,block2>>> (d_data);
    hipDeviceSynchronize();
    hipDeviceReset();


}

