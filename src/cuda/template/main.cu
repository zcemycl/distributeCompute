#include "hip/hip_runtime.h"
#include ""
#include "module.cuh"
#include <stdio.h>
#include <stdlib.h>

int main() {
    int size = 16;
    int byte_size = sizeof(int)*size;
    int h_data[] = {23,9,4,53,65,12,1,33,10,11,50,49,38,5,2,7};
    for (int i=0;i<size;i++){
        printf("%d ",h_data[i]);
    }
    printf("\n");

    int * d_data;
    hipMalloc((void**)&d_data,byte_size);
    hipMemcpy(d_data,h_data,byte_size,hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2);
    unique_gid_calc_2d <<<grid,block>>> (d_data);
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}