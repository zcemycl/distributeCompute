#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_cuda(){
    printf("Hello CUDA world \n");
}

void play_hello_cuda() {
    hello_cuda <<<1,20>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();
}