#include "hip/hip_runtime.h"
#include "ag_sum_arr/module.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <cstring>

__global__ void sum_arr_gpu(int * a, int * b, int * c, int size){
    // 1d grid, 1d block, all along x
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < size)
        c[gid] = a[gid] + b[gid];

}

void play_sum_arr_gpu(){
    int size = 10000;
    int block_size = 128;

    int NO_BYTES = size * sizeof(int);

    // host pointer
    int *h_a, *h_b, *gpu_results;
    h_a = (int*)malloc(NO_BYTES);
    h_b = (int*)malloc(NO_BYTES);
    gpu_results = (int*)malloc(NO_BYTES);

    time_t t;
    srand((unsigned)time(&t));
    for (int i=0; i<size; i++){
        h_a[i] = (int)(rand() & 0xFF);
        if (i<=10)
            std::cout << h_a[i] << " ";
    }
    std::cout << std::endl;
    for (int i=0; i<size; i++){
        h_b[i] = (int)(rand() & 0xFF);
        if (i<=10)
            std::cout << h_b[i] << " ";
    }
    std::cout << std::endl;
    memset(gpu_results, 0, NO_BYTES);

    // device pointer
    int *d_a, *d_b, *d_c;
    hipMalloc((int **)&d_a, NO_BYTES);
    hipMalloc((int **)&d_b, NO_BYTES);
    hipMalloc((int **)&d_c, NO_BYTES);

    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

    //launch the grid
    dim3 block(block_size);
    dim3 grid((size/block.x) + 1);

    sum_arr_gpu <<<grid,block>>> (d_a, d_b, d_c, size);
    hipDeviceSynchronize();

    hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);

    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);

    for (int i=0; i<11; i++){
        std::cout << gpu_results[i] << " ";
    }
    std::cout << std::endl;

    free(gpu_results);
    free(h_a);
    free(h_b);

    hipDeviceReset();

}