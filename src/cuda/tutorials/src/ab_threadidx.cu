#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "ab_threadidx/module.cuh"

__global__ void print_threadidx() {
    printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d\n",
        threadIdx.x, threadIdx.y, threadIdx.z);
}

void play_threadidx(){
    int nx = 16, ny = 16;
    dim3 block(8,8);
    dim3 grid(nx/block.x, ny/block.y);

    print_threadidx <<<grid,block>>> ();

    hipDeviceSynchronize();
    hipDeviceReset();
}