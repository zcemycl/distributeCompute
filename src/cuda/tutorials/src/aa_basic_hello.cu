#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void hello_cuda(){
    printf("Hello CUDA world \n");
}

void play_hello_cuda() {
    std::cout << "aa. Hello Cuda ..." << std::endl;
    hello_cuda <<<1,4>>> ();
    hipDeviceSynchronize();

    std::cout << "grid, block call 3x2, 1x1, 1x1..." << std::endl;
    dim3 block(2);
    dim3 grid(3);
    hello_cuda <<<grid,block>>> ();
    hipDeviceSynchronize();

    std::cout << "grid, block call 2x8, 2x2, 1x1..." << std::endl;
    dim3 blockex(8,2,1);
    dim3 gridex(2,2,1);
    hello_cuda <<<gridex,blockex>>> ();
    hipDeviceSynchronize();


    hipDeviceReset();
}