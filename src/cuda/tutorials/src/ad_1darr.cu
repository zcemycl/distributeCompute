#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ad_1darr/module.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void unique_idx_calc_threadIdx(int * input) {
    int tid = threadIdx.x;
    printf("threadIdx : %d, value : %d\n", tid, input[tid]);
}

void access_1d_arr() {
    int array_size = 8;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23,9,4,53,65,12,1,33}; // host data
    
    for (int i=0; i<array_size; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    int * d_data; // device data
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(8);
    dim3 grid(1);

    unique_idx_calc_threadIdx <<<grid,block>>> (d_data);
    hipDeviceSynchronize();

    hipDeviceReset();
}