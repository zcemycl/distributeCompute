#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "aj_warp_divergence/module.cuh"

__global__ void code_without_divergence() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    float a,b;
    a = b = 0;

    int warp_id = gid/32;

    if (warp_id % 2 == 0){
        a = 100.0;
        b = 50.0;
    } else {
        a = 200.0;
        b = 75.0;
    }
}

__global__ void divergence_code() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    float a,b;
    a = b = 0;

    if (gid % 2 == 0){
        a = 100.0;
        b = 50.0;
    } else {
        a = 200.0;
        b = 75.0;
    }
}


void test_warp_divergence(){
    printf("\n---------------------WARP DIVERGENCE EXAMPLE--------------------\n\n");

    int size = 1<<22;

    dim3 block(128);
    dim3 grid((size+block.x-1)/block.x);

    code_without_divergence <<<grid,block>>> ();
    hipDeviceSynchronize();

    divergence_code <<<grid,block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset(); 
}