#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "ai_warp_details/module.cuh"


__global__ void warp_details() {
    int gid = threadIdx.x + blockDim.x*blockIdx.x
        + blockDim.x*gridDim.x*blockIdx.y;
    int local_warp_id = threadIdx.x/32;
    int global_block_id = blockIdx.x + gridDim.x*blockIdx.y;
    int warp_x = blockDim.x/32 + 1;
    int global_warp_id = threadIdx.x/32 + warp_x*blockIdx.x 
        + warp_x*gridDim.x*blockIdx.y;

    printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, lwid : %d, gbid : %d, gwid : %d\n",
        threadIdx.x, blockIdx.x, blockIdx.y, gid, local_warp_id, global_block_id, global_warp_id);
}

void print_warp_details(){
    dim3 block(42);
    dim3 grid(2,2);

    warp_details <<<grid,block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();
}