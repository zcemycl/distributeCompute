#include "hip/hip_runtime.h"
#include "ag_sum_arr/module.cuh"
#include "common.hpp"
#include "common.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <cstring>

__global__ void sum_arr_gpu(int * a, int * b, int * c, int size){
    // 1d grid, 1d block, all along x
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < size)
        c[gid] = a[gid] + b[gid];

}

void sum_arr_cpu(int * a, int * b, int * c, int size){
    for (int i=0; i<size; i++)
        c[i] = a[i] + b[i];
}

void play_sum_arr_gpu(){
    hipError_t error;
    int size = 10000;
    int block_size = 128;

    int NO_BYTES = size * sizeof(int);

    // host pointer
    int *h_a, *h_b, *gpu_results, *h_c;
    h_a = (int*)malloc(NO_BYTES);
    h_b = (int*)malloc(NO_BYTES);
    gpu_results = (int*)malloc(NO_BYTES);
    h_c = (int*)malloc(NO_BYTES);

    time_t t;
    srand((unsigned)time(&t));
    for (int i=0; i<size; i++){
        h_a[i] = (int)(rand() & 0xFF);
        if (i<=10)
            std::cout << h_a[i] << " ";
    }
    std::cout << std::endl;
    for (int i=0; i<size; i++){
        h_b[i] = (int)(rand() & 0xFF);
        if (i<=10)
            std::cout << h_b[i] << " ";
    }
    std::cout << std::endl;
    clock_t cpu_start, cpu_end;
    cpu_start = clock();
    sum_arr_cpu(h_a, h_b, h_c, size);
    cpu_end = clock();
    memset(gpu_results, 0, NO_BYTES);

    // device pointer
    int *d_a, *d_b, *d_c;
    error = hipMalloc((int **)&d_a, NO_BYTES);
    if (error!=hipSuccess)
        fprintf(stderr, "Error : %s\n", hipGetErrorString(error));
    gpuErrchk(hipMalloc((int **)&d_b, NO_BYTES));
    hipMalloc((int **)&d_c, NO_BYTES);

    clock_t h2d_start, h2d_end;
    h2d_start = clock();
    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
    h2d_end = clock();

    //launch the grid
    dim3 block(block_size);
    dim3 grid((size/block.x) + 1);

    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum_arr_gpu <<<grid,block>>> (d_a, d_b, d_c, size);
    hipDeviceSynchronize();
    gpu_end = clock();

    clock_t d2h_start, d2h_end;
    d2h_start = clock();
    hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
    d2h_end = clock();

    compare_arrs(h_c, gpu_results, size);
    printf("Sum array CPU execution time : %4.6f \n", 
        (double)(double)(cpu_end-cpu_start)/CLOCKS_PER_SEC);
    printf("Sum array GPU execution time : %4.6f \n", 
        (double)(double)(gpu_end-gpu_start)/CLOCKS_PER_SEC);
    printf("htod mem transfer time : %4.6f \n", 
        (double)(double)(h2d_end-h2d_start)/CLOCKS_PER_SEC);
    printf("dtoh mem transfer time : %4.6f \n", 
        (double)(double)(d2h_end-d2h_start)/CLOCKS_PER_SEC);
    printf("Sum array GPU total execution time : %4.6f \n", 
        (double)(double)(d2h_end-h2d_start)/CLOCKS_PER_SEC);

    hipFree(d_c);
    hipFree(d_b);
    hipFree(d_a);

    for (int i=0; i<11; i++){
        std::cout << gpu_results[i] << " ";
    }
    std::cout << std::endl;

    free(gpu_results);
    free(h_a);
    free(h_b);

    hipDeviceReset();

}