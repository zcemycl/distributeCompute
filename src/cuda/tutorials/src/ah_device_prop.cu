#include "hip/hip_runtime.h"
#include "ah_device_prop/module.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>

void query_device(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
        printf("No CUDA support device found\n");
    
    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);

    printf("Device %d: %s\n", devNo, iProp.name);
    printf("  Number of multiprocessors: %d\n", iProp.multiProcessorCount);
    printf("  Clock rate: %d\n", iProp.clockRate);
    printf("  Compute capability: %d.%d\n", iProp.major, iProp.minor);
    printf("  Total amount of global memory: %4.2f MB\n", iProp.totalGlobalMem/pow(1024., 2.));
    printf("  Total amount of constant memory: %4.2f KB\n", iProp.totalConstMem/1024.0);
    printf("  Total amount of shared memory per block: %4.2f KB\n", iProp.sharedMemPerBlock/1024.0);
    printf("  Total amount of shared memory per MP: %4.2f KB\n", iProp.sharedMemPerMultiprocessor/1024.0);
    printf("  Total number of registers available per block: %d\n", iProp.regsPerBlock);
    printf("  Warp size: %d\n", iProp.warpSize);
    printf("  Maximum number of threads per block: %d\n", iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor: %d\n", iProp.maxThreadsPerMultiProcessor);
    // printf("  Maximum number of warps per multiprocessor: %d\n");
    printf("  Maximum Grid size: (%d,%d,%d)\n", iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
    printf("  Maximum block dimension: (%d,%d,%d)\n", iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
}