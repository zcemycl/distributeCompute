#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "af_3darr/module.cuh"

__global__ void unique_gid_calc_block2d_grid3d(int * data){
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int num_threads_in_block = blockDim.x * blockDim.y;
    int block_offset = blockIdx.x * num_threads_in_block;
    int num_threads_in_row = num_threads_in_block * gridDim.x;
    int row_offset = num_threads_in_row * blockIdx.y;
    int num_threads_in_xy = num_threads_in_row * gridDim.y;
    int xy_offset = num_threads_in_xy *  blockIdx.z;

    int gid = xy_offset + row_offset + block_offset + tid;
    printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, threadIdx.x : %d, threadIdx.z : %d, gid : %d - data : %d \n",
        blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, gid, data[gid]);

}

void access_3d_arr() {
    int array_size = 64;
    int array_byte_size = sizeof(int) * array_size;
    int * h_data;
    h_data = (int*)malloc(array_byte_size);

    time_t t;
    srand((unsigned)time(&t));
    for (int i=0; i<array_size; i++) {
        h_data[i] = rand() & 0xff;
    }

    int * d_data; // device data
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(2,2);
    dim3 grid(2,2,4);

    unique_gid_calc_block2d_grid3d <<<grid,block>>> (d_data);
    hipDeviceSynchronize();

    hipDeviceReset();
}