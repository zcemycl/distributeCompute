#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ad_1darr/module.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void unique_idx_calc_threadIdx(int * input) {
    int tid = threadIdx.x;
    printf("threadIdx : %d, value : %d\n", tid, input[tid]);
}

__global__ void unique_gid_calc(int * input) {
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;
    printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n", 
        blockIdx.x, tid, gid, input[gid]);
}

void access_1d_arr() {
    int array_size = 8;
    int array_byte_size = sizeof(int) * array_size;
    int h_data[] = {23,9,4,53,65,12,1,33}; // host data
    
    for (int i=0; i<array_size; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    int * d_data; // device data
    hipMalloc((void**)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

    dim3 block(8);
    dim3 grid(1);

    unique_idx_calc_threadIdx <<<grid,block>>> (d_data);
    hipDeviceSynchronize();

    dim3 block_4x2(4);
    dim3 grid_4x2(2);

    unique_gid_calc <<<grid_4x2,block_4x2>>> (d_data);
    hipDeviceSynchronize();
    

    hipDeviceReset();
}